// -*- mode:c++;tab-width:2;indent-tabs-mode:t;show-trailing-whitespace:t;rm-trailing-spaces:t -*-
// vi: set ts=2 noet:
//
// (c) Copyright Rosetta Commons Member Institutions.
// (c) This file is part of the Rosetta software suite and is made available under license.
// (c) The Rosetta software is developed by the contributing members of the Rosetta Commons.
// (c) For more information, see http://www.rosettacommons.org. Questions about this can be
// (c) addressed to University of Washington UW TechTransfer, email: license@u.washington.edu.

/// @file     core/scoring/sc/ShapeComplementarityCalculator_CPUKernels.cu
/// @brief    GPU Kernel code. This file needs to be compiled with NVIDIA's nvcc
/// @detailed Lawrence & Coleman shape complementarity calculator (based on CCP4's sc)
/// @author   Luki Goldschmidt <luki@mbi.ucla.edu>

/// This code was ported from the original Fortran code found in CCP4:
/// Sc (Version 2.0): A program for determining Shape Complementarity
/// Copyright Michael Lawrence, Biomolecular Research Institute
/// 343 Royal Parade Parkville Victoria Australia
///
/// This version contains support for GPU-acceleration by CUDA-capable devices,
/// which provides a 10-25x speed up over the CPU-only code using a regular desktop
/// video card with 4 processors (32 cores). Define CUDA_CPU and compile with


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define MIN(a,b) ((a) < (b) ? (a): (b))

//////////////////////////////////////////////////////////////////////
// Collision checking GPU kernel for TrimPeripheralBand

__global__ void _cuda_TrimPeripheralBand_kernel(
	float3 *dAccDotCoords,
	uint nAcc,
	float3 *dBurDotCoords,
	char *dDotColl,
	float r2)
{
	register int i, j, l;
	register float3 dot1;
	__shared__ char sColl[1024];
	__shared__ float3 sCoords[1024];

	sColl[threadIdx.x] = 0;
 	dot1 = dBurDotCoords[blockIdx.x*blockDim.x + threadIdx.x];

	for(i = 0; i < nAcc; i += blockDim.x) {
		__syncthreads();
		sCoords[threadIdx.x] = dAccDotCoords[i + threadIdx.x];
		__syncthreads();

		l = MIN(nAcc - i, blockDim.x);
		for(j = 0; j < l; j++) {
			register float3 dot2 = sCoords[j];
			dot2.x -= dot1.x;
			dot2.y -= dot1.y;
			dot2.z -= dot1.z;
			sColl[threadIdx.x] |= (dot2.x*dot2.x + dot2.y*dot2.y + dot2.z*dot2.z) <= r2;
		}
	}
	dDotColl[blockIdx.x*blockDim.x + threadIdx.x] = sColl[threadIdx.x];
}

//////////////////////////////////////////////////////////////////////
// Finding closest dot neighbor GPU kernel

__global__ void _cuda_FindClosestNeighbor_kernel(
	float3 *dMyDotCoords,
	float3 *dTheirDotCoords,
	uint nTheirDots,
	uint *dNeighbors)
{
	register int i, j, l;
	register float3 dot1;
	__shared__ uint sNeighbors[512];
	__shared__ float3 sCoords[512];
	float distmin = 99999.0, d2;

 	dot1 = dMyDotCoords[blockIdx.x*blockDim.x + threadIdx.x];

	for(i = 0; i < nTheirDots; i += blockDim.x) {
		__syncthreads();
		sCoords[threadIdx.x] = dTheirDotCoords[i + threadIdx.x];
		__syncthreads();

		l = MIN(nTheirDots - i, blockDim.x);
		for(j = 0; j < l; j++) {
			register float3 dot2 = sCoords[j];
			dot2.x -= dot1.x;
			dot2.y -= dot1.y;
			dot2.z -= dot1.z;
			d2 = dot2.x*dot2.x + dot2.y*dot2.y + dot2.z*dot2.z;
			if(d2 <= distmin) {
				distmin = d2;
				sNeighbors[threadIdx.x] = i+j;
			}
		}
	}
	dNeighbors[blockIdx.x*blockDim.x + threadIdx.x] = sNeighbors[threadIdx.x];
}

//////////////////////////////////////////////////////////////////////
// Stubs called from CPU code

void _cuda_sccalc_TrimPeripheralBand(int x, int y, float3 *dAccDotCoords, uint nAcc, float3 *dBurDotCoords, char *dDotColl, float r2)
{
	_cuda_TrimPeripheralBand_kernel<<<x, y>>>(dAccDotCoords, nAcc, dBurDotCoords, dDotColl, r2);
}

void _cuda_sccalc_FindClosestNeighbor(int x, int y, float3 *dMyDotCoords, float3 *dTheirDotCoords, uint nTheirDotsCoords, uint *dNeighbors)
{
	_cuda_FindClosestNeighbor_kernel<<<x, y>>>(dMyDotCoords, dTheirDotCoords, nTheirDotsCoords, dNeighbors);
}

